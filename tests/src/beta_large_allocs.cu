#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 * ============================================================================
 */





#include <poggers/counter_blocks/beta.cuh>

#include <poggers/beta/timer.cuh>


#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <chrono>

using namespace beta::allocators;


#if BETA_DEBUG_PRINTS
   #define TEST_BLOCK_SIZE 256
#else
   #define TEST_BLOCK_SIZE 256
#endif



template <typename allocator>
__global__ void large_alloc_kernel(allocator * alloc){


   uint64_t tid = poggers::utils::get_tid();

   if (tid != 0) return;


   void * allocation = nullptr;

   void * old_allocation = nullptr;



   for (uint64_t i = 1; i < 4096; i++){


      old_allocation = allocation;

      
      allocation = alloc->malloc(i*1024*1024);

      printf("Address allocated is %llx\n", (uint64_t) allocation);

      if (old_allocation != nullptr){
         alloc->free(old_allocation);
      }
      

   }

   //release last one
   alloc->free(allocation);

  

}


//pull from blocks
//this kernel tests correctness, and outputs misses in a counter.
template <uint64_t mem_segment_size, uint64_t smallest, uint64_t largest>
__host__ void beta_large_allocs(uint64_t num_bytes){


   beta::utils::timer boot_timing;

   using betta_type = beta::allocators::beta_allocator<mem_segment_size, smallest, largest>;


   betta_type * allocator = betta_type::generate_on_device(num_bytes, 111);

   std::cout << "Init in " << boot_timing.sync_end() << " seconds" << std::endl;

   beta::utils::timer kernel_timing;
   large_alloc_kernel<betta_type><<<1,1>>>(allocator);
   kernel_timing.sync_end();

   allocator->print_info();

   betta_type::free_on_device(allocator);

   hipDeviceSynchronize();

}



//using allocator_type = buddy_allocator<0,0>;

int main(int argc, char** argv) {


   //one_boot_betta_test_all_sizes<16ULL*1024*1024, 16ULL, 16ULL>(num_segments*16*1024*1024);  


   //beta_test_allocs_correctness<16ULL*1024*1024, 16ULL, 4096ULL>(num_segments*16*1024*1024, num_rounds, size);


   beta_large_allocs<16ULL*1024*1024, 16ULL, 4096ULL>(16ULL*1024*1024*1024);

   //beta_full_churn<16ULL*1024*1024, 16ULL, 4096ULL>(1600ULL*16*1024*1024,  num_segments, num_rounds);


   //beta_pointer_churn<16ULL*1024*1024, 16ULL, 4096ULL>(1600ULL*16*1024*1024,  num_segments, num_rounds);


   //beta_churn_no_free<16ULL*1024*1024, 16ULL, 4096ULL>(1600ULL*16*1024*1024,  num_segments);



   hipDeviceReset();
   return 0;

}

#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 * ============================================================================
 */





#include <poggers/counter_blocks/beta.cuh>

#include <poggers/beta/timer.cuh>


#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <chrono>


#define gpuErrorCheck(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

using namespace beta::allocators;


// __global__ void test_kernel(veb_tree * tree, uint64_t num_removes, int num_iterations){


//    uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

//    if (tid >= num_removes)return;


//       //printf("Tid %lu\n", tid);


//    for (int i=0; i< num_iterations; i++){


//       if (!tree->remove(tid)){
//          printf("BUG\n");
//       }

//       tree->insert(tid);

//    }




template <uint64_t mem_segment_size, uint64_t smallest>
__host__ void boot_alloc_table(){


   using table_type = alloc_table<mem_segment_size, smallest>;

   table_type * table = table_type::generate_on_device();

   hipDeviceSynchronize();

   table_type::free_on_device(table);

}
// }

// __global__ void view_kernel(veb_tree * tree){

//    uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

//    if (tid != 0)return;



// }

template <typename betta_type>
__global__ void register_all_segments(betta_type * betta, uint64_t num_segments){

   uint64_t tid = threadIdx.x + blockIdx.x * blockDim.x;

   if (tid >= num_segments) return;

   betta->gather_new_segment(0);

}


template <typename betta_type>
__global__ void malloc_all_blocks_single_thread(betta_type * betta, uint64_t num_segments, uint64_t blocks_per_segment){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid > 0) return; 

   uint64_t misses = 0;


   for (uint64_t i =0; i < num_segments; i++){

      printf("%llu/%llu\n", i, num_segments);

      for (uint64_t j = 0; j < blocks_per_segment; j++){

         Block * new_block = betta->table->get_block(i);

         if (new_block == nullptr){

            atomicAdd((unsigned long long int *)misses, 1);

         }

      }
   }

   printf("Total alloc misses: %llu/%llu\n", misses, num_segments*blocks_per_segment);


}

template <typename betta_type>
__global__ void malloc_all_blocks(betta_type * betta, uint64_t num_segments, uint64_t blocks_per_segment){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= num_segments*blocks_per_segment) return; 

   uint64_t segment_id = tid/blocks_per_segment;

   bool last_block;

   Block * new_block = betta->table->get_block(segment_id, last_block);

   if (new_block == nullptr){

      printf("Missed block %llu in section %llu\n", tid, segment_id);

   }




}

//pull all blocks using betta
template <typename betta_type>
__global__ void malloc_all_blocks_betta(betta_type * betta, uint64_t num_segments, uint64_t blocks_per_segment){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= num_segments*blocks_per_segment) return;

   Block * new_block = betta->request_new_block_from_tree(0);

   if (new_block == nullptr){
      printf("Failed to get block!\n");
   }

}


template <typename betta_type>
__global__ void malloc_and_save_blocks(betta_type * betta, Block ** blocks, uint64_t num_segments, uint64_t blocks_per_segment){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= num_segments*blocks_per_segment) return;

   Block * new_block = betta->request_new_block_from_tree(0);

   if (new_block == nullptr){
      printf("Alloc failure in 1\n");

      new_block = betta->request_new_block_from_tree(0);
   }

   blocks[tid] = new_block;

}


template <typename betta_type>
__global__ void malloc_and_save_blocks_tree(betta_type * betta, Block ** blocks, uint64_t num_segments, uint64_t blocks_per_segment, int tree_id){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= num_segments*blocks_per_segment) return;

   Block * new_block = betta->request_new_block_from_tree(tree_id);

   while (new_block == nullptr){
      //printf("Alloc failure in 2\n");

      new_block = betta->request_new_block_from_tree(tree_id);
   }

   blocks[tid] = new_block;

}


template <typename betta_type>
__global__ void betta_free_all_blocks(betta_type * betta, Block ** blocks, uint64_t num_segments, uint64_t blocks_per_segment){


   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= num_segments*blocks_per_segment) return;

   Block * new_block = blocks[tid];

   if (new_block == nullptr) return;

   betta->free_block(new_block);

}



template <typename betta_type>
__global__ void malloc_all_blocks_betta_single_thread(betta_type * betta, uint64_t num_segments, uint64_t blocks_per_segment){


   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid != 0) return;

   uint64_t misses = 0;

   for (uint64_t i = 0; i < num_segments*blocks_per_segment; i++){


      Block * new_block = betta->request_new_block_from_tree(0);

      if (new_block == nullptr){
         //printf("Failed to get block!\n");
         misses+=1;
      }

   }

   printf("Missed %llu/%llu\n", misses, num_segments*blocks_per_segment);

}


template <typename betta_type>
__global__ void peek(betta_type * betta){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid !=0 ) return;
}


template <typename betta_type>
__global__ void peek_blocks(betta_type * betta, Block ** blocks){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid !=0 ) return;
}


__global__ void assert_unique_blocks(Block ** blocks, uint64_t num_segments, uint64_t blocks_per_segment){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= num_segments*blocks_per_segment) return;

   uint64_t my_block = (uint64_t) blocks[tid];

   //if (my_block == 0) return;


   for (uint64_t i=tid+1; i < num_segments*blocks_per_segment; i++){

      uint64_t ext_block = (uint64_t) blocks[i];

      if (ext_block == my_block){
         printf("Collision on %llu and %llu: %llx\n", tid, i, ext_block);
      }

   }


}


template <typename betta_type>
__global__ void alloc_random_blocks(betta_type * betta){

   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   Block * my_blocks[10];

   uint64_t num_trees = betta_type::get_num_trees();


   for (int i = 0; i < 1; i++){

      int tree = poggers::hashers::MurmurHash64A (&tid, sizeof(uint64_t), i) % num_trees;

      my_blocks[i] = betta->request_new_block_from_tree(tree);

   }


   for (int i = 0; i < 1; i++){

      if (my_blocks[i] == nullptr){
         printf("Failed to alloc\n");
      } else {
         betta->free_block(my_blocks[i]);
      }

      

   }

   //printf("Done with %llu\n", tid);



}


// template <typename betta_type>
// __global__ void malloc_all_bits( )

// template <typename betta_type>
// __global__ void malloc_all_segments(betta_type * betta, uint64_t num_segments){

//    uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

//    if (tid >= num_segments) return;

//    betta

// }


template <uint64_t mem_segment_size, uint64_t smallest, uint64_t largest>
__host__ void boot_betta(uint64_t num_bytes){

   using betta_type = beta::allocators::beta_allocator<mem_segment_size, smallest, largest>;

   betta_type * allocator = betta_type::generate_on_device(num_bytes, 42);

   hipDeviceSynchronize();

   uint64_t num_segments = poggers::utils::get_max_chunks<mem_segment_size>(num_bytes);

   register_all_segments<betta_type><<<(num_segments-1)/512+1,512>>>(allocator, num_segments);

   printf("Ext sees %llu segments\n", num_segments);
   hipDeviceSynchronize();

   poggers::utils::print_mem_in_use();


   hipDeviceSynchronize();

   //malloc_all_blocks_single_thread<betta_type><<<1,1>>>(allocator, num_segments, 256);
   //malloc_all_blocks<betta_type><<<(num_segments*128-1)/512+1,512>>>(allocator, num_segments*128);

   malloc_all_blocks_betta<betta_type><<<(num_segments*256-1)/512+1,512>>>(allocator, num_segments, 256);

   hipDeviceSynchronize();

   peek<betta_type><<<1,1>>>(allocator);

   hipDeviceSynchronize();

   betta_type::free_on_device(allocator);

}



template <uint64_t mem_segment_size, uint64_t smallest, uint64_t largest>
__host__ void boot_betta_malloc_free(uint64_t num_bytes){

   using betta_type = beta::allocators::beta_allocator<mem_segment_size, smallest, largest>;

   betta_type * allocator = betta_type::generate_on_device(num_bytes, 42);

   hipDeviceSynchronize();

   uint64_t num_segments = poggers::utils::get_max_chunks<mem_segment_size>(num_bytes);

   register_all_segments<betta_type><<<(num_segments-1)/512+1,512>>>(allocator, num_segments);

   Block ** blocks;

   hipMalloc((void **)&blocks, sizeof(Block *)*num_segments*256);

   printf("Ext sees %llu segments\n", num_segments);
   hipDeviceSynchronize();

   poggers::utils::print_mem_in_use();


   hipDeviceSynchronize();

   //malloc_all_blocks_single_thread<betta_type><<<1,1>>>(allocator, num_segments, 256);
   //malloc_all_blocks<betta_type><<<(num_segments*128-1)/512+1,512>>>(allocator, num_segments*128);

   malloc_and_save_blocks<betta_type><<<(num_segments*256-1)/512+1,512>>>(allocator, blocks, num_segments, 256);

   hipDeviceSynchronize();

   allocator->print_info();

   hipDeviceSynchronize();


   assert_unique_blocks<<<(num_segments*256 -1)/512+1, 512>>>(blocks, num_segments, 256);

   peek_blocks<betta_type><<<1,1>>>(allocator, blocks);

   hipDeviceSynchronize();

   betta_free_all_blocks<betta_type><<<(num_segments*256-1)/512+1,512>>>(allocator, blocks, num_segments, 256);
   hipDeviceSynchronize();


   allocator->print_info();

   hipDeviceSynchronize();

   hipFree(blocks);

   betta_type::free_on_device(allocator);

}


template <uint64_t mem_segment_size, uint64_t smallest, uint64_t largest>
__host__ void boot_betta_test_all_sizes(uint64_t num_bytes){

   using betta_type = beta::allocators::beta_allocator<mem_segment_size, smallest, largest>;

   uint64_t num_trees = betta_type::get_num_trees();

   uint64_t num_segments = poggers::utils::get_max_chunks<mem_segment_size>(num_bytes);


   for (int i = 0; i< num_trees; i++){

      printf("Testing tree %d/%llu\n", i, num_trees);

      uint64_t blocks_per_segment = betta_type::get_blocks_per_segment(i);

      betta_type * allocator = betta_type::generate_on_device(num_bytes, 42);

      Block ** blocks;

      hipMalloc((void **)&blocks, sizeof(Block *)*num_segments*blocks_per_segment);

      hipDeviceSynchronize();

      printf("Boot done: allocator should be empty\n");
      allocator->print_info();


      hipDeviceSynchronize();

      malloc_and_save_blocks_tree<betta_type><<<(num_segments*blocks_per_segment-1)/512+1,512>>>(allocator, blocks, num_segments, blocks_per_segment, i);


      hipDeviceSynchronize();

      printf("Should see 0 free\n");
      allocator->print_info();

      hipDeviceSynchronize();

      assert_unique_blocks<<<(num_segments*blocks_per_segment -1)/512+1, 512>>>(blocks, num_segments, blocks_per_segment);

      hipDeviceSynchronize();

      betta_free_all_blocks<betta_type><<<(num_segments*blocks_per_segment-1)/512+1,512>>>(allocator, blocks, num_segments, blocks_per_segment);
   
      hipDeviceSynchronize();

      printf("Should see all free\n");
      allocator->print_info();

      hipDeviceSynchronize();


      hipFree(blocks);

      betta_type::free_on_device(allocator);

   }

   hipDeviceSynchronize();

}


template <uint64_t mem_segment_size, uint64_t smallest, uint64_t largest>
__host__ void one_boot_betta_test_all_sizes(uint64_t num_bytes){

   using betta_type = beta::allocators::beta_allocator<mem_segment_size, smallest, largest>;

   uint64_t num_trees = betta_type::get_num_trees();

   uint64_t num_segments = poggers::utils::get_max_chunks<mem_segment_size>(num_bytes);

   betta_type * allocator = betta_type::generate_on_device(num_bytes, 42);


   for (int i = 0; i< num_trees; i++){

      printf("Testing tree %d/%llu\n", i, num_trees);

      uint64_t blocks_per_segment = betta_type::get_blocks_per_segment(i);

      Block ** blocks;

      hipMalloc((void **)&blocks, sizeof(Block *)*num_segments*blocks_per_segment);

      hipDeviceSynchronize();

      printf("Boot done: allocator should be empty\n");
      allocator->print_info();


      hipDeviceSynchronize();

      beta::utils::timer malloc_timing;
      malloc_and_save_blocks_tree<betta_type><<<(num_segments*blocks_per_segment-1)/512+1,512>>>(allocator, blocks, num_segments, blocks_per_segment, i);
      auto malloc_duration = malloc_timing.sync_end();

      hipDeviceSynchronize();

      printf("Should see 0 free\n");
      allocator->print_info();

      hipDeviceSynchronize();

      assert_unique_blocks<<<(num_segments*blocks_per_segment -1)/512+1, 512>>>(blocks, num_segments, blocks_per_segment);

      hipDeviceSynchronize();


      beta::utils::timer free_timing;
      betta_free_all_blocks<betta_type><<<(num_segments*blocks_per_segment-1)/512+1,512>>>(allocator, blocks, num_segments, blocks_per_segment);
      auto free_duration = free_timing.sync_end();  

      hipDeviceSynchronize();

      printf("Should see all free\n");
      allocator->print_info();

      hipDeviceSynchronize();


      uint64_t total_num_blocks = num_segments*blocks_per_segment;

      malloc_timing.print_throughput("Alloced", total_num_blocks);
      free_timing.print_throughput("Freed", total_num_blocks);


      hipFree(blocks);

     

   }

   hipDeviceSynchronize();

   betta_type::free_on_device(allocator);

}

template <uint64_t mem_segment_size, uint64_t smallest, uint64_t largest>
__host__ void betta_alloc_random(uint64_t num_bytes, uint64_t num_allocs){

   using betta_type = beta::allocators::beta_allocator<mem_segment_size, smallest, largest>;

   uint64_t num_trees = betta_type::get_num_trees();

   uint64_t num_segments = poggers::utils::get_max_chunks<mem_segment_size>(num_bytes);

   betta_type * allocator = betta_type::generate_on_device(num_bytes, 42);

   alloc_random_blocks<betta_type><<<(num_allocs-1)/512+1, 512>>>(allocator);


   hipDeviceSynchronize();

   allocator->print_info();

   hipDeviceSynchronize();

   betta_type::free_on_device(allocator);

}


template<typename allocator_type>
__global__ void alloc_one_size(allocator_type * allocator, uint64_t num_allocs, uint64_t size){


   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= num_allocs) return;


   uint64_t malloc = allocator->malloc(size);

   if (malloc == ~0ULL){
      printf("Fail\n");
   }


}


//allocate from blocks, and print failures.
template <uint64_t mem_segment_size, uint64_t smallest, uint64_t largest>
__host__ void beta_test_allocs(uint64_t num_bytes){


   beta::utils::timer boot_timing;

   using betta_type = beta::allocators::beta_allocator<mem_segment_size, smallest, largest>;

   uint64_t num_segments = poggers::utils::get_max_chunks<mem_segment_size>(num_bytes);

   uint64_t allocs_per_segment = mem_segment_size/largest;

   uint64_t num_allocs = allocs_per_segment*num_segments;

   printf("Starting test with %lu segments, %lu allocs per segment\n", num_segments, allocs_per_segment);

   betta_type * allocator = betta_type::generate_on_device(num_bytes, 42);

   //generate bitarry

   uint64_t num_bytes_bitarray = sizeof(uint64_t)*((num_allocs -1)/64+1);

   uint64_t * bits;

   hipMalloc((void **)&bits, num_bytes_bitarray);

   hipMemset(bits, 0, num_bytes_bitarray);




   std::cout << "Init in " << boot_timing.sync_end() << " seconds" << std::endl;


   beta::utils::timer kernel_timing;
   alloc_one_size<betta_type><<<(num_allocs-1)/512+1,512>>>(allocator, .5*num_allocs, largest);
   kernel_timing.sync_end();


   kernel_timing.print_throughput("Malloced", .5*num_allocs);





   betta_type::free_on_device(allocator);


}

template<typename allocator_type>
__global__ void alloc_one_size_correctness(allocator_type * allocator, uint64_t num_allocs, uint64_t size, uint64_t * bitarray, uint64_t * misses){


   uint64_t tid = threadIdx.x+blockIdx.x*blockDim.x;

   if (tid >= num_allocs) return;


   uint64_t malloc = allocator->malloc(size);


   uint64_t attempts = 0;

   while (malloc == ~0ULL && attempts < 250){
      malloc = allocator->malloc(size);
      attempts+=1;
   }

   if (malloc == ~0ULL){
      atomicAdd((unsigned long long int *)misses, 1ULL);
      return;
   }

   uint64_t high = malloc / 64;

   uint64_t low = malloc % 64;

   auto bitmask = SET_BIT_MASK(low);

   uint64_t bits = atomicOr((unsigned long long int *) &bitarray[high], (unsigned long long int) bitmask);

   if (bits & bitmask){
      printf("Double malloc bug in %llu: block %llu alloc %llu", malloc, malloc/4096, malloc % 4096);
   }

   __threadfence();


}



//pull from blocks
//this kernel tests correctness, and outputs misses in a counter.
template <uint64_t mem_segment_size, uint64_t smallest, uint64_t largest>
__host__ void beta_test_allocs_correctness(uint64_t num_bytes){


   beta::utils::timer boot_timing;

   using betta_type = beta::allocators::beta_allocator<mem_segment_size, smallest, largest>;

   uint64_t num_segments = poggers::utils::get_max_chunks<mem_segment_size>(num_bytes);

   uint64_t allocs_per_segment = mem_segment_size/largest;

   uint64_t num_allocs = allocs_per_segment*num_segments;

   printf("Starting test with %lu segments, %lu allocs per segment\n", num_segments, allocs_per_segment);


   betta_type * allocator = betta_type::generate_on_device(num_bytes, 42);



   //generate bitarry
   uint64_t num_bytes_bitarray = sizeof(uint64_t)*((num_allocs -1)/64+1);

   uint64_t * bits;

   hipMalloc((void **)&bits, num_bytes_bitarray);

   hipMemset(bits, 0, num_bytes_bitarray);


   uint64_t * misses;
   hipMallocManaged((void **)&misses, sizeof(uint64_t));

   hipDeviceSynchronize();

   misses[0] = 0;




   std::cout << "Init in " << boot_timing.sync_end() << " seconds" << std::endl;


   beta::utils::timer kernel_timing;
   alloc_one_size_correctness<betta_type><<<(num_allocs-1)/512+1,512>>>(allocator, .9*num_allocs, largest, bits, misses);
   kernel_timing.sync_end();


   kernel_timing.print_throughput("Malloced", .9*num_allocs);

   printf("Missed: %llu\n", misses[0]);

   hipDeviceSynchronize();

   allocator->print_info();

   hipFree(misses);

   hipFree(bits);





   betta_type::free_on_device(allocator);


}


//using allocator_type = buddy_allocator<0,0>;

int main(int argc, char** argv) {

   uint64_t num_segments;
   

   if (argc < 2){
      num_segments = 100;
   } else {
      num_segments = std::stoull(argv[1]);
   }




   //one_boot_betta_test_all_sizes<16ULL*1024*1024, 16ULL, 16ULL>(num_segments*16*1024*1024);  


   beta_test_allocs_correctness<16ULL*1024*1024, 16ULL, 16ULL>(num_segments*16*1024*1024);

   hipDeviceReset();
   return 0;

}

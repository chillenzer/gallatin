#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 * ============================================================================
 */





#include <gallatin/allocators/global_allocator.cuh>

#include <gallatin/allocators/timer.cuh>

#include <gallatin/data_structs/extendible_ht.cuh>


#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <chrono>

using namespace gallatin::allocators;


#if GALLATIN_DEBUG_PRINTS
   #define TEST_BLOCK_SIZE 256
#else
   #define TEST_BLOCK_SIZE 256
#endif


// template <typename ht_type>
// __global__ void init_ht_kernel(ht_type * table, uint64_t num_slots, uint64_t seed, double resize_ratio){

//    uint64_t tid = gallatin::utils::get_tid();

//    if (tid != 0) return;

//    table->init(num_slots, seed, resize_ratio);

// }

// template <typename ht_type>
// __global__ void insert_kernel(ht_type * table, uint64_t ninserts){

//    uint64_t tid = gallatin::utils::get_tid();

//    if (tid >= ninserts) return;

//    table->insert(tid+1, tid);

// }


// template <typename ht_type>
// __global__ void insert_kernel_single(ht_type * table, uint64_t ninserts){

//    uint64_t tid = gallatin::utils::get_tid();

//    if (tid != 0) return;

//    for (uint i = 0; i < ninserts; i++){
//       table->insert(i+1, i);
//    }
   

// }

// template <typename ht_type, typename Val>
// __global__ void query_kernel(ht_type * table, uint64_t ninserts){

//    uint64_t tid = gallatin::utils::get_tid();

//    if (tid >= ninserts) return;

//    Val temp_val;

//    if (!table->query(tid+1, temp_val)){

//       printf("%lu Failed to query %lu\n", tid, tid+1);


//       table->query(tid+1, temp_val);

//    }

// }

// template <typename Key, typename Val, int stride = 1>
// __host__ void gallatin_ht_noresize(uint64_t num_bytes, uint64_t num_inserts, double init_fill_ratio, double resize_ratio){


//    using ht_type = gallatin::data_structs::quad_table<Key, Val, stride>;

//    gallatin::utils::timer boot_timing;

//    init_global_allocator(num_bytes, 42, false);


//    ht_type * table;
//    hipMalloc((void **)&table, sizeof(ht_type));

//    if (table == nullptr){
//       printf("Failed to malloc table\n");
//       free_global_allocator();
//       return;

//    }


//    init_ht_kernel<ht_type><<<1,1>>>(table, num_inserts*init_fill_ratio, 42, resize_ratio);

//    hipDeviceSynchronize();

//    //generate bitarry
//    //space reserved is one 

//    // uint64_t * misses;
//    // hipMallocManaged((void **)&misses, sizeof(uint64_t));

//    // hipDeviceSynchronize();

//    // misses[0] = 0;

//    hipDeviceSynchronize();

//    boot_timing.sync_end();

//    boot_timing.print_throughput("Booted", 1);

//    //and start kernel

//    gallatin::utils::timer insert_timing;

//    insert_kernel<ht_type><<<(num_inserts-1)/ TEST_BLOCK_SIZE +1, TEST_BLOCK_SIZE>>>(table, num_inserts);

//    //insert_kernel_single<ht_type><<<(num_inserts-1)/ TEST_BLOCK_SIZE +1, TEST_BLOCK_SIZE>>>(table, num_inserts);


//    insert_timing.sync_end();

//    gallatin::utils::timer query_timing;

//    query_kernel<ht_type, Val><<<(num_inserts-1)/ TEST_BLOCK_SIZE +1, TEST_BLOCK_SIZE>>>(table, num_inserts);

//    query_timing.sync_end();



//    insert_timing.print_throughput("Inserted", num_inserts);
//    query_timing.print_throughput("Queried", num_inserts);

//    free_global_allocator();


// }


template <typename ht>
__global__ void insert_ht_kernel(ht * table, uint64_t nitems){

   uint64_t tid = gallatin::utils::get_tid();

   if (tid >= nitems) return;

   table->insert(tid, tid);

}


template <typename Key, Key defaultKey, typename Val, int num_slots, uint64_t min_size, uint64_t max_size>
__host__ void extendible_ht_test(uint64_t num_bytes, double init_fill_ratio, double resize_ratio){


   using ht_type = gallatin::data_structs::extendible_hash_table<Key, defaultKey, Val, num_slots, min_size, max_size>;


   init_global_allocator(num_bytes, 42, false);

   auto my_table = ht_type::generate_on_device();

   // gallatin::utils::timer boot_timing;

   


   // ht_type * table;
   // hipMalloc((void **)&table, sizeof(ht_type));

   // if (table == nullptr){
   //    printf("Failed to malloc table\n");
   //    free_global_allocator();
   //    return;

   // }

   uint64_t nitems = max_size*.9;

   hipDeviceSynchronize();


   gallatin::utils::timer insert_timing;


   insert_ht_kernel<<<(nitems-1)/256+1, 256>>>(my_table, nitems);

   insert_timing.sync_end();

   insert_timing.print_throughput("Inserted", max_size);

   // init_ht_kernel<ht_type><<<1,1>>>(table, num_inserts*init_fill_ratio, 42, resize_ratio);

   // hipDeviceSynchronize();

   // //generate bitarry
   // //space reserved is one 

   // // uint64_t * misses;
   // // hipMallocManaged((void **)&misses, sizeof(uint64_t));

   // // hipDeviceSynchronize();

   // // misses[0] = 0;

   // hipDeviceSynchronize();

   // boot_timing.sync_end();

   // boot_timing.print_throughput("Booted", 1);

   // //and start kernel

   // gallatin::utils::timer insert_timing;

   // insert_kernel<ht_type><<<(num_inserts-1)/ TEST_BLOCK_SIZE +1, TEST_BLOCK_SIZE>>>(table, num_inserts);

   // //insert_kernel_single<ht_type><<<(num_inserts-1)/ TEST_BLOCK_SIZE +1, TEST_BLOCK_SIZE>>>(table, num_inserts);


   // insert_timing.sync_end();

   // gallatin::utils::timer query_timing;

   // query_kernel<ht_type, Val><<<(num_inserts-1)/ TEST_BLOCK_SIZE +1, TEST_BLOCK_SIZE>>>(table, num_inserts);

   // query_timing.sync_end();



   // insert_timing.print_throughput("Inserted", num_inserts);
   // query_timing.print_throughput("Queried", num_inserts);

   hipDeviceSynchronize();

   free_global_allocator();


}



int main(int argc, char** argv) {

   uint64_t num_segments;

   uint64_t num_inserts;


   if (argc < 2){
      num_segments = 1000;
   } else {
      num_segments = std::stoull(argv[1]);
   }

   if (argc < 3){
      num_inserts = 1000000;
   } else {
      num_inserts = std::stoull(argv[2]);
   }


   // printf(".2 .77\n");
   // gallatin_ht_noresize<uint64_t, uint64_t>(num_segments*16*1024*1024, num_inserts, .2, .77);


   // printf(".4 .77\n");
   // //gallatin_ht_noresize<uint64_t, uint64_t>(num_segments*16*1024*1024, num_inserts, .4, .77);

   // printf(".8 .77\n");
   // gallatin_ht_noresize<uint64_t, uint64_t>(num_segments*16*1024*1024, num_inserts, .8, .77);


   // printf(".2 .5\n");
   // gallatin_ht_noresize<uint64_t, uint64_t>(num_segments*16*1024*1024, num_inserts, .2, .5);

   // printf(".4 .5\n");
   // gallatin_ht_noresize<uint64_t, uint64_t>(num_segments*16*1024*1024, num_inserts, .4, .5);

   // printf(".8 .5\n");
   // gallatin_ht_noresize<uint64_t, uint64_t>(num_segments*16*1024*1024, num_inserts, .8, .5);



   //printf("Stride 0\n");
   extendible_ht_test<uint64_t, 0ULL, uint64_t, 15, 256, 256>(num_segments*16*1024*1024, .4, .5);

   // printf("Stride 1\n");
   // gallatin_ht_noresize<uint64_t, 0ULL, uint64_t, 1>(num_segments*16*1024*1024, .4, .5);

   // printf("Stride 2\n");
   // gallatin_ht_noresize<uint64_t, 0ULL, uint64_t, 2>(num_segments*16*1024*1024, .4, .5);

   // printf("Stride 8\n");
   // gallatin_ht_noresize<uint64_t, uint64_t, 8>(num_segments*16*1024*1024, num_inserts, .4, .5);

   //gallatin_ht_noresize<uint64_t, uint64_t, 3>(num_segments*16*1024*1024, num_inserts, 2, .77);

   //gallatin_ht_noresize<uint32_t, uint32_t, 4>(num_segments*16*1024*1024, num_inserts, 2, .77);

   hipDeviceSynchronize();

   hipDeviceReset();
   return 0;

}

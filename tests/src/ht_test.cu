#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 * ============================================================================
 */





#include <gallatin/allocators/global_allocator.cuh>

#include <gallatin/allocators/timer.cuh>

#include <gallatin/data_structs/quad_table_atomic.cuh>


#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <chrono>

using namespace gallatin::allocators;


#if GALLATIN_DEBUG_PRINTS
   #define TEST_BLOCK_SIZE 256
#else
   #define TEST_BLOCK_SIZE 256
#endif


template <typename ht_type>
__global__ void init_ht_kernel(ht_type * table, uint64_t num_slots, uint64_t seed){

   uint64_t tid = gallatin::utils::get_tid();

   if (tid != 0) return;

   table->init(num_slots, seed);

}

template <typename ht_type>
__global__ void insert_kernel(ht_type * table, uint64_t ninserts){

   uint64_t tid = gallatin::utils::get_tid();

   if (tid >= ninserts) return;

   table->insert(tid+1, tid);

}


template <typename ht_type>
__global__ void insert_kernel_single(ht_type * table, uint64_t ninserts){

   uint64_t tid = gallatin::utils::get_tid();

   if (tid != 0) return;

   for (uint i = 0; i < ninserts; i++){
      table->insert(i+1, i);
   }
   

}

template <typename ht_type, typename Val>
__global__ void query_kernel(ht_type * table, uint64_t ninserts){

   uint64_t tid = gallatin::utils::get_tid();

   if (tid >= ninserts) return;

   Val temp_val;

   if (!table->query(tid+1, temp_val)){

      printf("%lu Failed to query %lu\n", tid, tid+1);

   }

}

template <typename Key, typename Val>
__host__ void gallatin_ht_noresize(uint64_t num_bytes, uint64_t num_inserts){


   using ht_type = gallatin::data_structs::quad_table<Key, Val>;

   gallatin::utils::timer boot_timing;

   init_global_allocator(num_bytes, 42);


   ht_type * table;
   hipMalloc((void **)&table, sizeof(ht_type));

   if (table == nullptr){
      printf("Failed to malloc table\n");
      free_global_allocator();
      return;

   }


   init_ht_kernel<ht_type><<<1,1>>>(table, num_inserts*.5, 42);

   hipDeviceSynchronize();

   //generate bitarry
   //space reserved is one 

   // uint64_t * misses;
   // hipMallocManaged((void **)&misses, sizeof(uint64_t));

   // hipDeviceSynchronize();

   // misses[0] = 0;

   hipDeviceSynchronize();

   boot_timing.sync_end();

   boot_timing.print_throughput("Booted", 1);

   //and start kernel

   gallatin::utils::timer insert_timing;

   insert_kernel<ht_type><<<(num_inserts-1)/ TEST_BLOCK_SIZE +1, TEST_BLOCK_SIZE>>>(table, num_inserts);

   //insert_kernel_single<ht_type><<<(num_inserts-1)/ TEST_BLOCK_SIZE +1, TEST_BLOCK_SIZE>>>(table, num_inserts);


   insert_timing.sync_end();

   gallatin::utils::timer query_timing;

   query_kernel<ht_type, Val><<<(num_inserts-1)/ TEST_BLOCK_SIZE +1, TEST_BLOCK_SIZE>>>(table, num_inserts);

   query_timing.sync_end();



   insert_timing.print_throughput("Inserted", num_inserts);
   query_timing.print_throughput("Queried", num_inserts);

   free_global_allocator();


}



int main(int argc, char** argv) {

   uint64_t num_segments;

   uint64_t num_inserts;


   if (argc < 2){
      num_segments = 1000;
   } else {
      num_segments = std::stoull(argv[1]);
   }

   if (argc < 3){
      num_inserts = 1000000;
   } else {
      num_inserts = std::stoull(argv[2]);
   }

   gallatin_ht_noresize<uint64_t, uint64_t>(num_segments*16*1024*1024, num_inserts);

   hipDeviceSynchronize();

   hipDeviceReset();
   return 0;

}

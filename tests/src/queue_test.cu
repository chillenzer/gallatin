#include "hip/hip_runtime.h"
/*
 * ============================================================================
 *
 *        Authors:  
 *                  Hunter McCoy <hjmccoy@lbl.gov
 *
 * ============================================================================
 */




#include <poggers/allocators/alloc_utils.cuh>
#include <poggers/counter_blocks/beta.cuh>

#include <poggers/counter_blocks/timer.cuh>

#include <poggers/data_structs/custring.cuh>


#include <stdio.h>
#include <iostream>
#include <assert.h>
#include <chrono>

using namespace beta::allocators;

using namespace gallatin::data_structs;


//enqueue test kernel loads nitems into the queue, with every item unique based on TID
//then dequeue tests correctness by mapping to bitarry.
template <typename queue> 
__global__ void enqueue_test_kernel(queue * dev_queue, uint64_t nitems){

   uint64_t tid = poggers::utils::get_tid();

   if (tid >= nitems) return;

   dev_queue->enqueue(tid);
   
}


template <typename queue>
__global__ void dequeue_test_kernel(queue * dev_queue, uint64_t * bitarray, uint64_t nitems){

   uint64_t tid = poggers::utils::get_tid();

   if (tid >= nitems) return;

   uint64_t ext_tid;

   if (!dev_queue->dequeue(ext_tid)){
      printf("Thread %llu\n failed to dequeue item...\n", tid);
      return;
   }

   //all items dequeued, let's check correctness

   uint64_t high = ext_tid / 64;

   uint64_t low = ext_tid % 64;

   auto bitmask = SET_BIT_MASK(low);

   uint64_t bits = atomicOr((unsigned long long int *) &bitarray[high], (unsigned long long int) bitmask);

   if (bits & bitmask){
      printf("Double dequeue bug in %llu: block %llu alloc %llu\n", ext_tid, ext_tid/4096, ext_tid % 4096);
   }

}


__host__ void queue_test(uint64_t n_threads){

   using gallatin_allocator = beta::allocators::beta_allocator<16ULL*1024*1024, 16ULL, 4096ULL>;

   using queue_type = queue<uint64_t, gallatin_allocator>;

   //boot with 20 Gigs
   gallatin_allocator * alloc = gallatin_allocator::generate_on_device(20ULL*1024*1024*1024, 111);

   queue_type * dev_queue = queue_type::generate_on_device(alloc);


   uint64_t num_bytes_bitarray = sizeof(uint64_t)*((n_threads -1)/64+1);

   uint64_t * bits;

   hipMalloc((void **)&bits, num_bytes_bitarray);

   hipMemset(bits, 0, num_bytes_bitarray);

   hipDeviceSynchronize();

   printf("Starting queue test\n");

   beta::utils::timer enqueue_timing;

   enqueue_test_kernel<queue_type><<<(n_threads-1)/256 +1, 256>>>(dev_queue, n_threads);

   enqueue_timing.sync_end();

   enqueue_timing.print_throughput("Enqueued", n_threads);

   beta::utils::timer dequeue_timing;

   dequeue_test_kernel<queue_type><<<(n_threads-1)/256 +1, 256>>>(dev_queue, bits, n_threads);

   dequeue_timing.sync_end();

   

   dequeue_timing.print_throughput("Dequeued", n_threads);


}



//using allocator_type = buddy_allocator<0,0>;

int main(int argc, char** argv) {

   uint64_t num_threads;

   if (argc < 2){
      num_threads = 100;
   } else {
      num_threads = std::stoull(argv[1]);
   }


   queue_test(num_threads);

   hipDeviceReset();
   return 0;

}
